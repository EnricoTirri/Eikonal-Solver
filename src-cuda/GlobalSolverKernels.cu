#include "hip/hip_runtime.h"
#include "GlobalSolverKernels.hpp"
#include "LocalSolverKernels.hpp"
#include "iostream"

namespace Eikonal {

#define MAXF 900000.0;

    void allocateAndTransfer(void **dev_ptr, void *host_ptr, unsigned int type_size, unsigned int elem_number) {
        hipMalloc((void **) &(*dev_ptr), type_size * elem_number);
        hipMemcpy(*dev_ptr, host_ptr, type_size * elem_number, hipMemcpyHostToDevice);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) throw;
    }

    __global__ void
    initTimeReductionList(double *time_reduction_list, int list_size) {
        int blockSize = blockDim.x * blockDim.y;
        int blockId = blockIdx.y * gridDim.x + blockIdx.x;
        int threadId = threadIdx.y * blockDim.x + threadIdx.x;

        int id = blockId * blockSize + threadId;

        if (id < list_size) {
            time_reduction_list[id] = MAXF;
        }
    }

    __global__ void
    activateNeighbours(int *patchPatchPtr, int *patchAdjPatchIdx, int *converged_patch_list,
                       int *converged_patch_list_new, int n_patches) {
        int blockSize = blockDim.x * blockDim.y;
        int blockId = blockIdx.y * gridDim.x + blockIdx.x;

        int threadId = threadIdx.y * blockDim.x + threadIdx.x;

        int id = blockId * blockSize + threadId;

        if (id < n_patches) {
            if (converged_patch_list[threadId]) {
                int pRS = patchPatchPtr[threadId];
                int pRE = patchPatchPtr[threadId + 1];

                for (int i = pRS; i < pRE; ++i) {
                    converged_patch_list_new[patchAdjPatchIdx[i]] = 1;
                }
            }
        }
    }

    __global__ void
    scanAndPack(int *converged_activePatchList, int *activePatchList, int *activePatchList_size, int n_patches) {
        int threadId = threadIdx.y * blockDim.x + threadIdx.x;

        if (threadId < n_patches) {
            //PARALLEL SCAN FOR ACTIVE LIST PACK INDICES

            int stride = 1;
            for (; stride < n_patches; stride *= 2) {
                int step = 2 * stride;

                int k = step - 1 + threadId * step;
                if (k < n_patches) {
                    converged_activePatchList[k] += converged_activePatchList[k - stride];
                }

                __syncthreads();
            }

            stride = stride / 2;

            for (; stride > 1; stride /= 2) {
                int step = stride / 2;

                int k = stride - 1 + threadId * stride;
                if (k < n_patches - step) {
                    converged_activePatchList[k + step] += converged_activePatchList[k];
                }

                __syncthreads();
            }


            if (converged_activePatchList[threadId] > 0) {
                if (threadId == 0 || converged_activePatchList[threadId - 1] < converged_activePatchList[threadId]) {
                    activePatchList[converged_activePatchList[threadId] - 1] = threadId;
                }
            }

            if (threadId == 0)
                *activePatchList_size = converged_activePatchList[n_patches - 1];
        }
    }

    __global__ void
    updatePatchValues3(double *U, MprimeMatrix<3> *MprimeList, const int iterations,
                       const int *activePatchList, const int *activePatchList_size,
                       const int *patchElementPtr, const int *patchAdjElementIdx,
                       const int *patchNodePtr, const int *patchAdjNodeIdx,
                       const int *elementNodePtr, const int *elementAdjNodeIdx,
                       double *time_reduction_list, const int *time_reduction_ptr,
                       const int reductionSpan, int *converged_patch_list) {

        // GET BLOCK ID == ACTIVE PATCH POS
        int blockId = blockIdx.y * gridDim.x + blockIdx.x;

        // CHECK IF BLOCK ID IS IN RANGE OF ACTIVE PATCHES
        if (blockId < *activePatchList_size) {
            // GET ACTIVE PATCH ID
            int patchId = activePatchList[blockId];

            // GET THREAD ID == ELEMENT POS IN PATCH
            int threadId = threadIdx.y * blockDim.x + threadIdx.x;

            // GET NUMBER OF NODE AND ELEMENTS OF THE PATCH
            int patchEleStart = patchElementPtr[patchId];
            int patchEleEnd = patchElementPtr[patchId + 1];
            int patchEleSize = patchEleEnd - patchEleStart;

            int patchNodeStart = patchNodePtr[patchId];
            int patchNodeEnd = patchNodePtr[patchId + 1];
            int patchNodeSize = patchNodeEnd - patchNodeStart;

            // RETRIEVE MAX BETWEEN N_ELEMENT AND N_NODES (needed to dispatch threads among elements and nodes)
            int maxn = patchEleSize > patchNodeSize ? patchEleSize : patchNodeSize;

            // IF THREAD IS IN RANGE OF MAX
            if (threadId < maxn) {

                int elePos;
                MprimeMatrix<3> MT;
                int eleId;
                int eleStart;
                int eleEnd;
                double times[3];

                // IF THREAD IS IN RANGE OF ELEMENTS
                if (threadId < patchEleSize) {
                    // get element position in per patch element sorted list
                    elePos = patchEleStart + threadId;

                    // get MprimeMatrix associated to element
                    MT = MprimeList[elePos];

                    // get element Id and range of points
                    eleId = patchAdjElementIdx[elePos];
                    eleStart = elementNodePtr[eleId];
                    eleEnd = elementNodePtr[eleId + 1];
                }

                int nodePos;
                int nodeId;

                // IF THREAD IS IN RANGE OF NODES
                if (threadId < patchNodeSize) {
                    // get node position in per path node sorted list
                    nodePos = patchNodeStart + threadId;
                    // get node id
                    nodeId = patchAdjNodeIdx[nodePos];
                    // init node as not converged
                    //converged_reduction_list[nodePos] = 0;
                }

                if (threadId == 0)
                    converged_patch_list[patchId] = 0;

                // MAIN ITERATION
                for (int iteration = 0; iteration < iterations; ++iteration) {

                    __syncthreads();

                    // IF THREAD IN RANGE OF ELEMENTS
                    if (threadId < patchEleSize) {

                        // RETRIEVE CURRENT TIMES OF VERTICES
                        for (int i = eleStart; i < eleEnd; ++i) {
                            // retrieve current time associated to each point
                            int pointId = elementAdjNodeIdx[i];
                            times[i - eleStart] = U[pointId];
                        }

                        // UPDATE ALL TIMES OF VERTICES
                        for (int i = 0; i < 3; ++i) {

                            // solve for each vertex of element
                            double sol = solveLocal<3>(i, MT, times);

                            // if locally converged
                            if (times[i] > sol) {
                                // locally assign solution
                                times[i] = sol;
                                // place result in the reduction list for later reconciliation of values
                                time_reduction_list[time_reduction_ptr[elePos * 3 + i]] = sol;
                            }
                        }
                    }


                    // WAIT FOR ALL THREADS TO HAVE CALCULATED TIMES OF ELEMENTS
                    __syncthreads();


                    // IF THREAD IN RANGE OF NODES => REDUCTION OF TIME ON NODES OF PATCH
                    if (threadId < patchNodeSize) {

                        // init reduction variable on first of reduction node values
                        int listPos = nodeId * reductionSpan;

                        double reducedValue = time_reduction_list[listPos];
                        // over other reduction node values
                        for (int i = 1; i < reductionSpan; ++i)
                            // apply reduction if i-th value is smaller
                            if (reducedValue > time_reduction_list[listPos + i])
                                reducedValue = time_reduction_list[listPos + i];


                        // check if node has converged
                        if (U[nodeId] > reducedValue) {
                            // set as converged and update node time value
                            converged_patch_list[patchId] = 1;
                            U[nodeId] = reducedValue;
                        }
                    }
                }
            }
        }
    }

    __global__ void
    updatePatchValues4(double *U, MprimeMatrix<4> *MprimeList, const int iterations,
                       const int *activePatchList, const int *activePatchList_size,
                       const int *patchElementPtr, const int *patchAdjElementIdx,
                       const int *patchNodePtr, const int *patchAdjNodeIdx,
                       const int *elementNodePtr, const int *elementAdjNodeIdx,
                       double *time_reduction_list, const int *time_reduction_ptr,
                       const int reductionSpan, int *converged_patch_list) {

        // GET BLOCK ID == ACTIVE PATCH POS
        int blockId = blockIdx.y * gridDim.x + blockIdx.x;

        // CHECK IF BLOCK ID IS IN RANGE OF ACTIVE PATCHES
        if (blockId < *activePatchList_size) {
            // GET ACTIVE PATCH ID
            int patchId = activePatchList[blockId];

            // GET THREAD ID == ELEMENT POS IN PATCH
            int threadId = threadIdx.y * blockDim.x + threadIdx.x;

            // GET NUMBER OF NODE AND ELEMENTS OF THE PATCH
            int patchEleStart = patchElementPtr[patchId];
            int patchEleEnd = patchElementPtr[patchId + 1];
            int patchEleSize = patchEleEnd - patchEleStart;

            int patchNodeStart = patchNodePtr[patchId];
            int patchNodeEnd = patchNodePtr[patchId + 1];
            int patchNodeSize = patchNodeEnd - patchNodeStart;

            // RETRIEVE MAX BETWEEN N_ELEMENT AND N_NODES (needed to dispatch threads among elements and nodes)
            int maxn = patchEleSize > patchNodeSize ? patchEleSize : patchNodeSize;

            // IF THREAD IS IN RANGE OF MAX
            if (threadId < maxn) {

                int elePos;
                MprimeMatrix<4> MT;
                int eleId;
                int eleStart;
                int eleEnd;
                double times[4];

                // IF THREAD IS IN RANGE OF ELEMENTS
                if (threadId < patchEleSize) {
                    // get element position in per patch element sorted list
                    elePos = patchEleStart + threadId;

                    // get MprimeMatrix associated to element
                    MT = MprimeList[elePos];

                    // get element Id and range of points
                    eleId = patchAdjElementIdx[elePos];
                    eleStart = elementNodePtr[eleId];
                    eleEnd = elementNodePtr[eleId + 1];
                }

                int nodePos;
                int nodeId;

                // IF THREAD IS IN RANGE OF NODES
                if (threadId < patchNodeSize) {
                    // get node position in per path node sorted list
                    nodePos = patchNodeStart + threadId;
                    // get node id
                    nodeId = patchAdjNodeIdx[nodePos];
                    // init node as not converged
                    //converged_reduction_list[nodePos] = 0;
                }

                if (threadId == 0)
                    converged_patch_list[patchId] = 0;

                // MAIN ITERATION
                for (int iteration = 0; iteration < iterations; ++iteration) {

                    __syncthreads();

                    // IF THREAD IN RANGE OF ELEMENTS
                    if (threadId < patchEleSize) {

                        // RETRIEVE CURRENT TIMES OF VERTICES
                        for (int i = eleStart; i < eleEnd; ++i) {
                            // retrieve current time associated to each point
                            int pointId = elementAdjNodeIdx[i];
                            times[i - eleStart] = U[pointId];
                        }

                        // UPDATE ALL TIMES OF VERTICES
                        for (int i = 0; i < 4; ++i) {

                            // solve for each vertex of element
                            double sol = solveLocal<4>(i, MT, times);

                            // if locally converged
                            if (times[i] > sol) {
                                // locally assign solution
                                times[i] = sol;
                                // place result in the reduction list for later reconciliation of values
                                time_reduction_list[time_reduction_ptr[elePos * 4 + i]] = sol;
                            }
                        }
                    }


                    // WAIT FOR ALL THREADS TO HAVE CALCULATED TIMES OF ELEMENTS
                    __syncthreads();


                    // IF THREAD IN RANGE OF NODES => REDUCTION OF TIME ON NODES OF PATCH
                    if (threadId < patchNodeSize) {

                        // init reduction variable on first of reduction node values
                        int listPos = nodeId * reductionSpan;

                        double reducedValue = time_reduction_list[listPos];
                        // over other reduction node values
                        for (int i = 1; i < reductionSpan; ++i)
                            // apply reduction if i-th value is smaller
                            if (reducedValue > time_reduction_list[listPos + i])
                                reducedValue = time_reduction_list[listPos + i];


                        // check if node has converged
                        if (U[nodeId] > reducedValue) {
                            // set as converged and update node time value
                            converged_patch_list[patchId] = 1;
                            U[nodeId] = reducedValue;
                        }
                    }
                }
            }
        }
    }

    void checkError(const std::string &message = "") {
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cout << hipGetErrorName(err) << " " << message << std::endl;
            throw;
        }
    }


    template<>
    void globalSolve<3>(const std::vector<int> &XPatches,
                        const std::vector<int> &patchElementPtr, const std::vector<int> &patchAdjElementIdx,
                        const std::vector<int> &patchNodePtr, const std::vector<int> &patchAdjNodeIdx,
                        const std::vector<int> &elementNodePtr, const std::vector<int> &elementAdjNodeIdx,
                        const std::vector<int> &patchPatchPtr, const std::vector<int> &patchAdjPatchIdx,
                        std::vector<double> &U, const std::vector<MprimeMatrix<3>> &MprimeList,
                        const std::vector<int> &time_reduction_ptr, const int reduction_span,
                        bool *result) {

        int block_size = BLOCK_SIZE;

        int n_patches = patchElementPtr.size() - 1;
        int n_elements = patchAdjElementIdx.size();
        int n_nodes = patchAdjNodeIdx.size();

        // INIT CUDA PATCH TO ELEMENT ADJACENT LIST
        int *patchElementPtr_dev, *patchAdjElementIdx_dev;
        {
            allocateAndTransfer((void **) &patchElementPtr_dev, (void *) patchElementPtr.data(),
                                sizeof(int), patchElementPtr.size());
            allocateAndTransfer((void **) &patchAdjElementIdx_dev, (void *) patchAdjElementIdx.data(),
                                sizeof(int), patchAdjElementIdx.size());
        }

        // INIT CUDA PATCH TO NODE ADJACENT LIST
        int *patchNodePtr_dev, *patchAdjNodeIdx_dev;
        {
            allocateAndTransfer((void **) &patchNodePtr_dev, (void *) patchNodePtr.data(),
                                sizeof(int), patchNodePtr.size());
            hipMemcpy((void *) patchNodePtr.data(), patchNodePtr_dev, sizeof(int) * patchNodePtr.size(),
                       hipMemcpyDeviceToHost);

            allocateAndTransfer((void **) &patchAdjNodeIdx_dev, (void *) patchAdjNodeIdx.data(),
                                sizeof(int), patchAdjNodeIdx.size());
        }

        // INIT CUDA ELEMENT TO NODE ADJACENT LIST
        int *elementNodePtr_dev, *elementAdjNodeIdx_dev;
        {
            allocateAndTransfer((void **) &elementNodePtr_dev, (void *) elementNodePtr.data(),
                                sizeof(int), elementNodePtr.size());
            allocateAndTransfer((void **) &elementAdjNodeIdx_dev, (void *) elementAdjNodeIdx.data(),
                                sizeof(int), elementAdjNodeIdx.size());
        }

        // INIT CUDA PATCH TO PATCH ADJACENT LIST
        int *patchPatchPtr_dev, *patchAdjPatchIdx_dev;
        {
            allocateAndTransfer((void **) &patchPatchPtr_dev, (void *) patchPatchPtr.data(),
                                sizeof(int), patchPatchPtr.size());
            allocateAndTransfer((void **) &patchAdjPatchIdx_dev, (void *) patchAdjPatchIdx.data(),
                                sizeof(int), patchAdjPatchIdx.size());
        }

        // INIT CUDA MPRIMEMATRIX PER ELEMENT LIST
        MprimeMatrix<3> *MprimeList_dev;
        {
            allocateAndTransfer((void **) &MprimeList_dev, (void *) MprimeList.data(),
                                sizeof(MprimeMatrix<3>), MprimeList.size());
        }

        // INIT CUDA TIME REDUCTION LIST AND POINTER LIST
        int *time_reduction_ptr_dev;
        double *time_reduction_list_dev;
        {
            allocateAndTransfer((void **) &time_reduction_ptr_dev, (void *) time_reduction_ptr.data(),
                                sizeof(int), time_reduction_ptr.size());

            // init all time reduction list at MAXF
            int list_size = reduction_span * n_nodes;
            hipMalloc((void **) &time_reduction_list_dev, sizeof(double) * list_size);
            int n_blocks = (list_size + block_size) / block_size;
            initTimeReductionList<<<n_blocks, block_size>>>(time_reduction_list_dev, list_size);
        }

        // INIT CUDA SUPPORT LISTS FOR CONVERGED NODES AND PATCHES
        int *converged_patch_list_dev, *converged_patch_list_new_dev;
        {
            hipMalloc((void **) &converged_patch_list_dev, sizeof(int) * n_patches);
            hipMalloc((void **) &converged_patch_list_new_dev, sizeof(int) * n_patches);
        }

        // INIT CUDA TIME
        double *U_dev;
        {
            allocateAndTransfer((void **) &U_dev, (void *) U.data(),
                                sizeof(double), U.size());
        }

        // INIT CUDA ACTIVE LIST AND SIZE
        std::vector<int> activePatchList(n_patches);
        std::copy(XPatches.begin(), XPatches.end(), activePatchList.data());

        int activeListSize = XPatches.size();
        int *activePatchList_dev, *activePatchListSize_dev;
        {
            allocateAndTransfer((void **) &activePatchList_dev, (void *) activePatchList.data(),
                                sizeof(int), n_patches);
            allocateAndTransfer((void **) &activePatchListSize_dev, (void *) &activeListSize,
                                sizeof(int), 1);
        }


        std::vector<int> convergedPatchList(n_patches);
        std::vector<int> convergedPatchList_new(n_patches);

        int n_blocks = (n_patches + block_size) / block_size;

        timespec start{}, end{};
        clock_gettime(CLOCK_MONOTONIC, &start);

        while (activeListSize > 0) {

            // UPDATE ACTIVE PATCH
            {
                hipMemset(converged_patch_list_dev, 0, sizeof(int) * n_patches);
                checkError("A");
                updatePatchValues3<<<activeListSize, block_size>>>(U_dev, MprimeList_dev, 7,
                                                                   activePatchList_dev, activePatchListSize_dev,
                                                                   patchElementPtr_dev, patchAdjElementIdx_dev,
                                                                   patchNodePtr_dev, patchAdjNodeIdx_dev,
                                                                   elementNodePtr_dev, elementAdjNodeIdx_dev,
                                                                   time_reduction_list_dev, time_reduction_ptr_dev,
                                                                   reduction_span, converged_patch_list_dev);

                checkError("B");
            }

            // ACTIVATE NEIGHBOURS OF CONVERGED PATCHES (MIXED)
            {
                hipMemset(converged_patch_list_new_dev, 0, sizeof(int) * n_patches);
                activateNeighbours<<<n_blocks, block_size>>>(patchPatchPtr_dev, patchAdjPatchIdx_dev, converged_patch_list_dev, converged_patch_list_new_dev, n_patches);
                hipMemcpy(convergedPatchList_new.data(), converged_patch_list_new_dev, sizeof(int) * n_patches, hipMemcpyDeviceToHost);

                int k = 0;
                for (int i = 0; i < convergedPatchList_new.size(); ++i)
                    if (convergedPatchList_new[i] > 0) {
                        activePatchList[k] = i;
                        k++;
                    }

                activeListSize = k;

                hipMemcpy(activePatchList_dev, activePatchList.data(), sizeof(int) * activeListSize,
                           hipMemcpyHostToDevice);
                checkError("D");
                hipMemcpy(activePatchListSize_dev, &activeListSize, sizeof(int), hipMemcpyHostToDevice);
                checkError("E");
            }


            if(activeListSize==0) break;

            //DO ONE ITERATION ON NEIGHBOURS
            {
                hipMemset(converged_patch_list_dev, 0, sizeof(int) * n_patches);
                checkError("F");

                updatePatchValues3<<<activeListSize, block_size>>>(U_dev, MprimeList_dev, 1,
                                                                   activePatchList_dev, activePatchListSize_dev,
                                                                   patchElementPtr_dev, patchAdjElementIdx_dev,
                                                                   patchNodePtr_dev, patchAdjNodeIdx_dev,
                                                                   elementNodePtr_dev, elementAdjNodeIdx_dev,
                                                                   time_reduction_list_dev, time_reduction_ptr_dev,
                                                                   reduction_span, converged_patch_list_dev);
                checkError("G");
            }

            //RETRIEVE ACTIVE LIST (CPU)
            {
                hipMemcpy(convergedPatchList.data(), converged_patch_list_dev, sizeof(int) * n_patches,
                           hipMemcpyDeviceToHost);
                checkError("H");

                int k = 0;
                for (int i = 0; i < convergedPatchList_new.size(); ++i)
                    if (convergedPatchList_new[i] > 0) {
                        activePatchList[k] = i;
                        k++;
                    }

                activeListSize = k;

                hipMemcpy(activePatchList_dev, activePatchList.data(), sizeof(int) * n_patches,
                           hipMemcpyHostToDevice);
                checkError("I");
                hipMemcpy(activePatchListSize_dev, &activeListSize, sizeof(int), hipMemcpyHostToDevice);
                checkError("L");
            }

        }

        // GET FINAL VALUES
        hipMemcpy(U.data(), U_dev, sizeof(double) * U.size(), hipMemcpyDeviceToHost);

        clock_gettime(CLOCK_MONOTONIC, &end);

        auto elapsed = static_cast<double>((end.tv_sec - start.tv_sec));
        elapsed += static_cast<double>((end.tv_nsec - start.tv_nsec)) / 1000000000.0;
        std::cout << "pure computational time: " << elapsed << std::endl;

        // FREE ALL CUDA MEMORY
        {
            hipFree(patchElementPtr_dev);
            hipFree(patchAdjElementIdx_dev);
            hipFree(patchNodePtr_dev);
            hipFree(patchAdjNodeIdx_dev);
            hipFree(elementNodePtr_dev);
            hipFree(elementAdjNodeIdx_dev);
            hipFree(patchPatchPtr_dev);
            hipFree(patchAdjPatchIdx_dev);
            hipFree(activePatchList_dev);
            hipFree(U_dev);
            hipFree(MprimeList_dev);
            hipFree(time_reduction_ptr_dev);
            hipFree(time_reduction_list_dev);
            hipFree(converged_patch_list_dev);
            hipFree(activePatchListSize_dev);
        }

        (*result) = true;
    }

    template<>
    void globalSolve<4>(const std::vector<int> &XPatches,
                        const std::vector<int> &patchElementPtr, const std::vector<int> &patchAdjElementIdx,
                        const std::vector<int> &patchNodePtr, const std::vector<int> &patchAdjNodeIdx,
                        const std::vector<int> &elementNodePtr, const std::vector<int> &elementAdjNodeIdx,
                        const std::vector<int> &patchPatchPtr, const std::vector<int> &patchAdjPatchIdx,
                        std::vector<double> &U, const std::vector<MprimeMatrix<4>> &MprimeList,
                        const std::vector<int> &time_reduction_ptr, const int reduction_span,
                        bool *result) {

        int block_size = BLOCK_SIZE;

        int n_patches = patchElementPtr.size() - 1;
        int n_elements = patchAdjElementIdx.size();
        int n_nodes = patchAdjNodeIdx.size();

        // INIT CUDA PATCH TO ELEMENT ADJACENT LIST
        int *patchElementPtr_dev, *patchAdjElementIdx_dev;
        {
            allocateAndTransfer((void **) &patchElementPtr_dev, (void *) patchElementPtr.data(),
                                sizeof(int), patchElementPtr.size());
            allocateAndTransfer((void **) &patchAdjElementIdx_dev, (void *) patchAdjElementIdx.data(),
                                sizeof(int), patchAdjElementIdx.size());
        }

        // INIT CUDA PATCH TO NODE ADJACENT LIST
        int *patchNodePtr_dev, *patchAdjNodeIdx_dev;
        {
            allocateAndTransfer((void **) &patchNodePtr_dev, (void *) patchNodePtr.data(),
                                sizeof(int), patchNodePtr.size());
            hipMemcpy((void *) patchNodePtr.data(), patchNodePtr_dev, sizeof(int) * patchNodePtr.size(),
                       hipMemcpyDeviceToHost);

            allocateAndTransfer((void **) &patchAdjNodeIdx_dev, (void *) patchAdjNodeIdx.data(),
                                sizeof(int), patchAdjNodeIdx.size());
        }

        // INIT CUDA ELEMENT TO NODE ADJACENT LIST
        int *elementNodePtr_dev, *elementAdjNodeIdx_dev;
        {
            allocateAndTransfer((void **) &elementNodePtr_dev, (void *) elementNodePtr.data(),
                                sizeof(int), elementNodePtr.size());
            allocateAndTransfer((void **) &elementAdjNodeIdx_dev, (void *) elementAdjNodeIdx.data(),
                                sizeof(int), elementAdjNodeIdx.size());
        }

        // INIT CUDA PATCH TO PATCH ADJACENT LIST
        int *patchPatchPtr_dev, *patchAdjPatchIdx_dev;
        {
            allocateAndTransfer((void **) &patchPatchPtr_dev, (void *) patchPatchPtr.data(),
                                sizeof(int), patchPatchPtr.size());
            allocateAndTransfer((void **) &patchAdjPatchIdx_dev, (void *) patchAdjPatchIdx.data(),
                                sizeof(int), patchAdjPatchIdx.size());
        }

        // INIT CUDA MPRIMEMATRIX PER ELEMENT LIST
        MprimeMatrix<4> *MprimeList_dev;
        {
            allocateAndTransfer((void **) &MprimeList_dev, (void *) MprimeList.data(),
                                sizeof(MprimeMatrix<4>), MprimeList.size());
        }

        // INIT CUDA TIME REDUCTION LIST AND POINTER LIST
        int *time_reduction_ptr_dev;
        double *time_reduction_list_dev;
        {
            allocateAndTransfer((void **) &time_reduction_ptr_dev, (void *) time_reduction_ptr.data(),
                                sizeof(int), time_reduction_ptr.size());

            // init all time reduction list at MAXF
            int list_size = reduction_span * n_nodes;
            hipMalloc((void **) &time_reduction_list_dev, sizeof(double) * list_size);
            int n_blocks = (list_size + block_size) / block_size;
            initTimeReductionList<<<n_blocks, block_size>>>(time_reduction_list_dev, list_size);
        }

        // INIT CUDA SUPPORT LISTS FOR CONVERGED NODES AND PATCHES
        int *converged_patch_list_dev, *converged_patch_list_new_dev;
        {
            hipMalloc((void **) &converged_patch_list_dev, sizeof(int) * n_patches);
            hipMalloc((void **) &converged_patch_list_new_dev, sizeof(int) * n_patches);
        }

        // INIT CUDA TIME
        double *U_dev;
        {
            allocateAndTransfer((void **) &U_dev, (void *) U.data(),
                                sizeof(double), U.size());
        }

        // INIT CUDA ACTIVE LIST AND SIZE
        std::vector<int> activePatchList(n_patches);
        std::copy(XPatches.begin(), XPatches.end(), activePatchList.data());

        int activeListSize = XPatches.size();
        int *activePatchList_dev, *activePatchListSize_dev;
        {
            allocateAndTransfer((void **) &activePatchList_dev, (void *) activePatchList.data(),
                                sizeof(int), n_patches);
            allocateAndTransfer((void **) &activePatchListSize_dev, (void *) &activeListSize,
                                sizeof(int), 1);
        }


        std::vector<int> convergedPatchList(n_patches);
        std::vector<int> convergedPatchList_new(n_patches);

        int n_blocks = (n_patches + block_size) / block_size;

        while (activeListSize > 0) {

            // UPDATE ACTIVE PATCH
            {
                hipMemset(converged_patch_list_dev, 0, sizeof(int) * n_patches);
                checkError("A");
                updatePatchValues4<<<activeListSize, block_size>>>(U_dev, MprimeList_dev, 7,
                                                                   activePatchList_dev, activePatchListSize_dev,
                                                                   patchElementPtr_dev, patchAdjElementIdx_dev,
                                                                   patchNodePtr_dev, patchAdjNodeIdx_dev,
                                                                   elementNodePtr_dev, elementAdjNodeIdx_dev,
                                                                   time_reduction_list_dev, time_reduction_ptr_dev,
                                                                   reduction_span, converged_patch_list_dev);

                checkError("B");
            }

            // ACTIVATE NEIGHBOURS OF CONVERGED PATCHES (MIXED)
            {
                hipMemset(converged_patch_list_new_dev, 0, sizeof(int) * n_patches);
                activateNeighbours<<<n_blocks, block_size>>>(patchPatchPtr_dev, patchAdjPatchIdx_dev, converged_patch_list_dev, converged_patch_list_new_dev, n_patches);
                hipMemcpy(convergedPatchList_new.data(), converged_patch_list_new_dev, sizeof(int) * n_patches, hipMemcpyDeviceToHost);

                int k = 0;
                for (int i = 0; i < convergedPatchList_new.size(); ++i)
                    if (convergedPatchList_new[i] > 0) {
                        activePatchList[k] = i;
                        k++;
                    }

                activeListSize = k;

                std::cout << k << std::endl;

                hipMemcpy(activePatchList_dev, activePatchList.data(), sizeof(int) * activeListSize,
                           hipMemcpyHostToDevice);
                checkError("D");
                hipMemcpy(activePatchListSize_dev, &activeListSize, sizeof(int), hipMemcpyHostToDevice);
                checkError("E");
            }


            if(activeListSize==0) break;

            //DO ONE ITERATION ON NEIGHBOURS
            {
                hipMemset(converged_patch_list_dev, 0, sizeof(int) * n_patches);
                checkError("F");

                updatePatchValues4<<<activeListSize, block_size>>>(U_dev, MprimeList_dev, 1,
                                                                   activePatchList_dev, activePatchListSize_dev,
                                                                   patchElementPtr_dev, patchAdjElementIdx_dev,
                                                                   patchNodePtr_dev, patchAdjNodeIdx_dev,
                                                                   elementNodePtr_dev, elementAdjNodeIdx_dev,
                                                                   time_reduction_list_dev, time_reduction_ptr_dev,
                                                                   reduction_span, converged_patch_list_dev);
                checkError("G");
            }

            //RETRIEVE ACTIVE LIST (CPU)
            {
                hipMemcpy(convergedPatchList.data(), converged_patch_list_dev, sizeof(int) * n_patches,
                           hipMemcpyDeviceToHost);
                checkError("H");

                int k = 0;
                for (int i = 0; i < convergedPatchList_new.size(); ++i)
                    if (convergedPatchList_new[i] > 0) {
                        activePatchList[k] = i;
                        k++;
                    }

                activeListSize = k;

                hipMemcpy(activePatchList_dev, activePatchList.data(), sizeof(int) * n_patches,
                           hipMemcpyHostToDevice);
                checkError("I");
                hipMemcpy(activePatchListSize_dev, &activeListSize, sizeof(int), hipMemcpyHostToDevice);
                checkError("L");
            }

        }

        // GET FINAL VALUES
        hipMemcpy(U.data(), U_dev, sizeof(double) * U.size(), hipMemcpyDeviceToHost);

        // FREE ALL CUDA MEMORY
        {
            hipFree(patchElementPtr_dev);
            hipFree(patchAdjElementIdx_dev);
            hipFree(patchNodePtr_dev);
            hipFree(patchAdjNodeIdx_dev);
            hipFree(elementNodePtr_dev);
            hipFree(elementAdjNodeIdx_dev);
            hipFree(patchPatchPtr_dev);
            hipFree(patchAdjPatchIdx_dev);
            hipFree(activePatchList_dev);
            hipFree(U_dev);
            hipFree(MprimeList_dev);
            hipFree(time_reduction_ptr_dev);
            hipFree(time_reduction_list_dev);
            hipFree(converged_patch_list_dev);
            hipFree(activePatchListSize_dev);
        }

        (*result) = true;
    }
}