#include "GlobalSolverKernels.hpp"
#include "iostream"

namespace Eikonal{

    void allocateAndTransfer(void **dev_ptr, void *host_ptr, unsigned int type_size, unsigned int elem_number){
        hipMalloc((void **)&(*dev_ptr), type_size * elem_number);
        hipMemcpy(*dev_ptr, host_ptr, type_size * elem_number, hipMemcpyHostToDevice);
    }

}