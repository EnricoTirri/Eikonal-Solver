#include "hip/hip_runtime.h"

#include "LocalSolverKernels.hpp"

namespace Eikonal {

    // EIKONAL MATH KERNELS --------------------------------------------------------------------------------
    __device__ inline bool sameSign(double a, double b) {
        return a * b >= 0;
    }

    __device__ inline double
    constraintBisection(double ma, double mb, double mc, double dt, const int max_iters, const double tol) {
        auto f = [ma, mb, mc, dt](const double x) {
            return (x * ma + mb) / std::sqrt(2 * x * mb + x * x * ma + mc) - dt;
        };

        double a = 0;
        double b = 1;
        double fa = f(a);
        double fb = f(b);
        if (sameSign(fa, fb)) {
            if (std::abs(fa) < std::abs(fb))
                return 0;
            else
                return 1;
        }
        double k;
        int iters = max_iters;
        double res;
        do {
            k = (a + b) / 2;
            res = f(k);
            if (sameSign(fa, res)) {
                a = k;
                fa = res;
            } else {
                b = k;
            }
            iters--;
        } while (std::abs(res) > tol && iters != 0);
        return k;
    }
    // -------------------------------------------------------------------------------------------------------



    // GLOBALLY USED LOCAL SOLVER FUNCTIONS ----------------------------------------------------------
    __device__ int esgn(const int &kcode, const int &lcode, const int &scode) {
        return (2 * (scode < kcode) - 1) * (2 * (scode < lcode) - 1);
    }

    __device__ int signParity(const int &graycode) {
        int s = graycode - ((graycode >> 1) & 033333333333) - ((graycode >> 2) & 011111111111);
        return 1 - 2 * ((((s + (s >> 3)) & 030707070707) % 63) % 2);
    }
    // -----------------------------------------------------------------------------------------------



    // TRIANGULAR LOCAL SOLVER KERNELS --------------------------------------------------------------
    __device__ inline void getSigns3(int *signs, const int &shift) {
        signs[0] = signParity((24 >> shift) & 7);
        signs[1] = signParity((16 >> shift) & 7);
    }


    __device__ void getMprimeMatrix3(const int &ptidx, const MprimeMatrix<3> &MT, const double *valin,
                                     MprimeMatrix<3> &M, double *valout) {
        constexpr int RED_SIZE = 3 - 1;
        int gcodes[RED_SIZE];
        int signs[RED_SIZE];

        // get gcodes (rotated of -1)
        int refId = (3 + ptidx - 1) % 3;
        int refcode = (1 << refId);

        for (int i = 0; i < RED_SIZE; ++i) {
            int k = (ptidx + i) % 3;
            gcodes[(RED_SIZE + i - 1) % RED_SIZE] = refcode + (1 << k);
        }

        // get rotated vals (rotated of shift)
        int shift = RED_SIZE - ptidx;
        for (int i = 0; i < RED_SIZE; ++i) {
            valout[i] = valin[(3 + i - shift) % 3];
        }

        getSigns3(signs, shift);

        int e0 = gcodes[0] / 2 - 1;
        M(0) = MT(e0);

        int e1 = gcodes[1] / 2 - 1;
        M(1) = MT(e1);

        int g2 = gcodes[0] xor gcodes[1];
        int s01 = esgn(gcodes[0], gcodes[1], g2);
        int e2 = g2 / 2 - 1;
        M(2) = s01 * signs[0] * signs[1] * (MT(e0) + MT(e1) - MT(e2)) / 2;
    }

    __device__ inline double distance3(const double l1, const MprimeMatrix<3> &M) {
        return std::sqrt(l1 * l1 * M(0) + 2 * l1 * M(2) + M(1));
    }


    template<>
    __device__ double solveLocal<3>(const int &pointref, const MprimeMatrix<3> &MT, double *valin) {
        TTraits<3>::MprimeMatrix M;
        double valout[3];

        getMprimeMatrix3(pointref, MT, valin, M, valout);

        double t12 = valout[1] - valout[0];
        double l1 = constraintBisection(M(0), M(2), M(1), t12, 2, 10e-6);
        return -t12 * l1 + valout[1] + distance3(l1, M);
    }

    // ------------------------------------------------------------------------------------------------


    // TETRAHEDRAL LOCAL SOLVER KERNELS ---------------------------------------------------------------
    __device__ inline void getSigns4(int *signs, const int &shift) {
        signs[0] = signParity((80 >> shift) & 15);
        signs[1] = signParity((96 >> shift) & 15);
        signs[2] = signParity((192 >> shift) & 15);
    }

    __device__ inline double distance4(const double l1, const double l2, const MprimeMatrix<4> &M) {
        return std::sqrt(l1 * l1 * M(0) + l2 * l2 * M(1) + 2 * (l1 * l2 * M(2) + l1 * M(5) + l2 * M(4)) + M(3));
    }


    __device__ void getMprimeMatrix4(const int &ptidx, const MprimeMatrix<4> &MT, const double *valin,
                                     MprimeMatrix<4> &M, double *valout) {

        constexpr int RED_SIZE = 4 - 1;
        int gcodes[RED_SIZE];
        int signs[RED_SIZE];

        // get gcodes (rotated of -1)
        int refId = (4 + ptidx - 1) % 4;
        int refcode = (1 << refId);

        for (int i = 0; i < RED_SIZE; ++i) {
            int k = (ptidx + i) % 4;
            gcodes[(RED_SIZE + i - 1) % RED_SIZE] = refcode + (1 << k);
        }

        // get rotated vals (rotated of shift)
        int shift = RED_SIZE - ptidx;
        for (int i = 0; i < RED_SIZE; ++i) {
            valout[i] = valin[(4 + i - shift) % 4];
        }

        getSigns4(signs, shift);

        int e0 = gcodes[0] / 2 - 1;
        M(0) = MT(e0);

        int e1 = gcodes[1] / 2 - 1;
        M(1) = MT(e1);

        int g2 = gcodes[0] xor gcodes[1];
        int s01 = esgn(gcodes[0], gcodes[1], g2);
        int e2 = g2 / 2 - 1;
        M(2) = s01 * signs[0] * signs[1] * (MT(e0) + MT(e1) - MT(e2)) / 2;

        int e3 = gcodes[2] / 2 - 1;
        M(3) = MT(e3);

        int g4 = gcodes[1] xor gcodes[2];
        int s12 = esgn(gcodes[1], gcodes[2], g4);
        int e4 = g4 / 2 - 1;
        M(4) = s12 * signs[1] * signs[2] * (MT(e1) + MT(e3) - MT(e4)) / 2;

        int g5 = gcodes[0] xor gcodes[2];
        int s02 = esgn(gcodes[0], gcodes[2], g5);
        int e5 = g5 / 2 - 1;
        M(5) = s02 * signs[0] * signs[2] * (MT(e0) + MT(e3) - MT(e5)) / 2;
    }


    template<>
    __device__ double solveLocal<4>(const int &pointref, const MprimeMatrix<4> &MT, double *valin) {

        TTraits<4>::MprimeMatrix M;
        double valout[4];
        getMprimeMatrix4(pointref, MT, valin, M, valout);

        using Vector = Eigen::Matrix<double, 2, 1>;
        using Jacobian = Eigen::Matrix<double, 2, 2>;
        double l1 = 0;
        double l2 = 0;
        const double t13 = valout[2] - valout[0];
        const double t23 = valout[2] - valout[1];
        Vector R;
        int iters = 2;
        do {
            double dist = distance4(l1, l2, M);
            double la = l1 * M(0) + l2 * M(2) + M(5);
            double lb = l1 * M(2) + l2 * M(1) + M(4);
            R << -t13 * dist + la, -t23 * dist + lb;
            Jacobian J;
            J.row(0) << M(1) - t23 * lb / dist,
                        -(M(2) - t13 * lb / dist);
            J.row(1) << -(M(2) - t23 * la / dist),
                    M(0) - t13 * la / dist,

            J /= (J(0,0)*J(1,1) - J(0,1) * J(1,0));

            Vector dir = -J * R;
            l1 += dir(0);
            l2 += dir(1);
            if (l1 <= 0) {
                l1 = 0;
                if (l2 <= 0) {
                    l2 = 0;
                } else {
                    l2 = constraintBisection(M(1), M(4), M(3), t23, 5000, 10e-6);
                }
                break;
            } else {
                if (l2 <= 0) {
                    l1 = constraintBisection(M(0), M(5), M(3), t13, 5000, 10e-6);
                    l2 = 0;
                    break;
                } else if (l1 + l2 >= 1) {
                    double p1 = M(0) - 2 * M(2) + M(1);
                    double p2 = M(2) + M(5) - M(1) - M(4);
                    double p3 = M(1) + 2 * M(4) + M(3);
                    l1 = constraintBisection(p1, p2, p3, t13 - t23, 5000, 10e-6);
                    l2 = 1 - l1;
                    break;
                }
            }
            iters--;
        } while (R.norm() > 10e-6 && iters != 0);
        return -t13 * l1 - t23 * l2 + valout[2] + distance4(l1, l2, M);
    }



    // ------------------------------------------------------------------------------------------------
}