#include "hip/hip_runtime.h"
#include "GlobalSolverKernels.hpp"
#include "LocalSolverKernels.hpp"
#include "iostream"

namespace Eikonal {

#define MAXF 900000.0;

    void allocateAndTransfer(void **dev_ptr, void *host_ptr, unsigned int type_size, unsigned int elem_number) {
        hipMalloc((void **) &(*dev_ptr), type_size * elem_number);
        hipMemcpy(*dev_ptr, host_ptr, type_size * elem_number, hipMemcpyHostToDevice);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) throw;
    }

    __global__ void
    initTimeReductionList(double *time_reduction_list, int list_size) {
        int blockSize = blockDim.x * blockDim.y;
        int blockId = blockIdx.y * gridDim.x + blockIdx.x;
        int threadId = threadIdx.y * blockDim.x + threadIdx.x;

        int id = blockId * blockSize + threadId;

        if (id < list_size) {
            time_reduction_list[id] = MAXF;
        }
    }


    __global__ void
    activateNeighbours(int *patchPatchPtr, int *patchAdjPatchIdx, int *activatedPatches, int n_patches) {
        int threadId = threadIdx.y * blockDim.x + threadIdx.x;

        if (threadId < n_patches) {
            int converged = activatedPatches[threadId];
            __syncthreads();

            if (converged) {
                int pRS = patchPatchPtr[threadId];
                int pRE = patchPatchPtr[threadId + 1];

                for (int i = pRS; i < pRE; ++i) {
                    activatedPatches[patchAdjPatchIdx[i]] = 1;
                }
            }
        }
    }

    __global__ void
    scanAndPack(int *converged_activePatchList, int *activePatchList, int *activePatchList_size, int n_patches) {
        int threadId = threadIdx.y * blockDim.x + threadIdx.x;

        if (threadId < n_patches) {
            //PARALLEL SCAN FOR ACTIVE LIST PACK INDICES

            int stride = 1;
            for (; stride < n_patches; stride *= 2) {
                int step = 2 * stride;

                int k = step - 1 + threadId * step;
                if (k < n_patches) {
                    converged_activePatchList[k] += converged_activePatchList[k - stride];
                }

                __syncthreads();
            }

            stride = stride / 2;

            for (; stride > 1; stride /= 2) {
                int step = stride / 2;

                int k = stride - 1 + threadId * stride;
                if (k < n_patches - step) {
                    converged_activePatchList[k + step] += converged_activePatchList[k];
                }

                __syncthreads();
            }


            if (converged_activePatchList[threadId] > 0) {
                if (threadId == 0 || converged_activePatchList[threadId - 1] < converged_activePatchList[threadId]) {
                    activePatchList[converged_activePatchList[threadId] - 1] = threadId;
                }
            }

            if (threadId == 0)
                *activePatchList_size = converged_activePatchList[n_patches - 1];
        }
    }

    __global__ void
    updatePatchValues3(double *U, MprimeMatrix<3> *MprimeList, const int iterations,
                       const int *activePatchList, const int *activePatchList_size,
                       const int *patchElementPtr, const int *patchAdjElementIdx,
                       const int *patchNodePtr, const int *patchAdjNodeIdx,
                       const int *elementNodePtr, const int *elementAdjNodeIdx,
                       double *time_reduction_list, const int *time_reduction_ptr, const int reductionSpan,
                       int *converged_reduction_list, int *converged_patch_list) {

        // GET BLOCK ID == ACTIVE PATCH POS
        int blockId = blockIdx.y * gridDim.x + blockIdx.x;

        // CHECK IF BLOCK ID IS IN RANGE OF ACTIVE PATCHES
        if (blockId < *activePatchList_size) {
            // GET ACTIVE PATCH ID
            int patchId = activePatchList[blockId];

            // GET THREAD ID == ELEMENT POS IN PATCH
            int threadId = threadIdx.y * blockDim.x + threadIdx.x;

            // GET NUMBER OF NODE AND ELEMENTS OF THE PATCH
            int patchEleStart = patchElementPtr[patchId];
            int patchEleEnd = patchElementPtr[patchId + 1];
            int patchEleSize = patchEleEnd - patchEleStart;

            int patchNodeStart = patchNodePtr[patchId];
            int patchNodeEnd = patchNodePtr[patchId + 1];
            int patchNodeSize = patchNodeEnd - patchNodeStart;

            // RETRIEVE MAX BETWEEN N_ELEMENT AND N_NODES (needed to dispatch threads among elements and nodes)
            int maxn = patchEleSize > patchNodeSize ? patchEleSize : patchNodeSize;

            // IF THREAD IS IN RANGE OF MAX
            if (threadId < maxn) {

                int elePos;
                MprimeMatrix<3> MT;
                int eleId;
                int eleStart;
                int eleEnd;
                double times[3];

                // IF THREAD IS IN RANGE OF ELEMENTS
                if (threadId < patchEleSize) {
                    // get element position in per patch element sorted list
                    elePos = patchEleStart + threadId;

                    // get MprimeMatrix associated to element
                    MT = MprimeList[elePos];

                    // get element Id and range of points
                    eleId = patchAdjElementIdx[elePos];
                    eleStart = elementNodePtr[eleId];
                    eleEnd = elementNodePtr[eleId + 1];
                }

                int nodePos;
                int nodeId;

                // IF THREAD IS IN RANGE OF NODES
                if (threadId < patchNodeSize) {
                    // get node position in per path node sorted list
                    nodePos = patchNodeStart + threadId;
                    // get node id
                    nodeId = patchAdjNodeIdx[nodePos];
                    U[nodeId] = nodePos;
                }

                return;
                // MAIN ITERATION
                for (int iteration = 0; iteration < iterations; ++iteration) {

                    __syncthreads();

                    // IF THREAD IN RANGE OF ELEMENTS
                    if (threadId < patchEleSize) {

                        // RETRIEVE CURRENT TIMES OF VERTICES
                        for (int i = eleStart; i < eleEnd; ++i) {
                            // retrieve current time associated to each point
                            int pointId = elementAdjNodeIdx[i];
                            times[i - eleStart] = U[pointId];
                        }

                        // UPDATE ALL TIMES OF VERTICES
                        for (int i = 0; i < 3; ++i) {

                            // solve for each vertex of element
                            double sol = solveLocal<3>(i, MT, times);

                            // if locally converged
                            if (times[i] > sol) {
                                // locally assign solution
                                times[i] = sol;
                                // place result in the reduction list for later reconciliation of values
                                time_reduction_list[time_reduction_ptr[elePos * 3 + i]] = sol;
                            }
                        }
                    }

                    // WAIT FOR ALL THREADS TO HAVE CALCULATED TIMES OF ELEMENTS
                    __syncthreads();


                    // IF THREAD IN RANGE OF NODES => REDUCTION OF TIME ON NODES OF PATCH
                    if (threadId < patchNodeSize) {
                        // init node ad not converged
                        converged_reduction_list[nodePos] = 0;

                        // init reduction variable on first of reduction node values
                        int listPos = nodeId * reductionSpan;
                        U[nodeId] = threadId;
                        return;

                        double reducedValue = time_reduction_list[listPos];
                        // over other reduction node values
                        for (int i = 1; i < reductionSpan; ++i)
                            // apply reduction if i-th value is smaller
                            if (reducedValue > time_reduction_list[listPos + i])
                                reducedValue = time_reduction_list[listPos + i];

                        // check if node has converged
                        if (U[nodeId] > reducedValue) {
                            // set as converged and update node time value
                            converged_reduction_list[nodePos] = 1;
                            U[nodeId] = reducedValue;
                        }
                    }
                }

                if (threadId == 0)
                    converged_patch_list[patchId] = patchNodeSize; //converged_reduction_list[0] && converged_reduction_list[1];
                return;

                // INIT PATCH AS NOT CONVERGED
                if (threadId == 0)
                    converged_patch_list[patchId] = 0;

                // WAIT FOR ALL THREAD TO HAVE APPLIED REDUCTION
                __syncthreads();

                for (unsigned int s = (patchNodeSize+1) / 2; s > 1; s = (s+1)/2) {
                    if (threadId <= s && threadId + s < patchNodeSize) {
                        converged_reduction_list[nodePos] &= converged_reduction_list[nodePos + s];
                    }
                    patchNodeSize = s;
                    __syncthreads();
                }

                if (threadId == 0)
                    converged_patch_list[patchId] = nodePos; //converged_reduction_list[0] && converged_reduction_list[1];
            }
        }
    }


    template<>
    void globalSolve<3>(const std::vector<int> &XPatches,
                        const std::vector<int> &patchElementPtr, const std::vector<int> &patchAdjElementIdx,
                        const std::vector<int> &patchNodePtr, const std::vector<int> &patchAdjNodeIdx,
                        const std::vector<int> &elementNodePtr, const std::vector<int> &elementAdjNodeIdx,
                        const std::vector<int> &patchPatchPtr, const std::vector<int> &patchAdjPatchIdx,
                        std::vector<double> &U, const std::vector<MprimeMatrix<3>> &MprimeList,
                        const std::vector<int> &time_reduction_ptr, const int reduction_span,
                        bool *result) {

        int block_size = 512;

        int n_patches = patchElementPtr.size() - 1;
        int n_elements = patchAdjElementIdx.size();
        int n_nodes = patchAdjNodeIdx.size();

        // INIT CUDA PATCH TO ELEMENT ADJACENT LIST
        int *patchElementPtr_dev, *patchAdjElementIdx_dev;
        {
            allocateAndTransfer((void **) &patchElementPtr_dev, (void *) patchElementPtr.data(),
                                sizeof(int), patchElementPtr.size());
            allocateAndTransfer((void **) &patchAdjElementIdx_dev, (void *) patchAdjElementIdx.data(),
                                sizeof(int), patchAdjElementIdx.size());
        }

        // INIT CUDA PATCH TO NODE ADJACENT LIST
        int *patchNodePtr_dev, *patchAdjNodeIdx_dev;
        {
            allocateAndTransfer((void **) &patchNodePtr_dev, (void *) patchNodePtr.data(),
                                sizeof(int), patchNodePtr.size());
            hipMemcpy((void *) patchNodePtr.data(), patchNodePtr_dev, sizeof(int) * patchNodePtr.size(), hipMemcpyDeviceToHost);

            allocateAndTransfer((void **) &patchAdjNodeIdx_dev, (void *) patchAdjNodeIdx.data(),
                                sizeof(int), patchAdjNodeIdx.size());
        }

        // INIT CUDA ELEMENT TO NODE ADJACENT LIST
        int *elementNodePtr_dev, *elementAdjNodeIdx_dev;
        {
            allocateAndTransfer((void **) &elementNodePtr_dev, (void *) elementNodePtr.data(),
                                sizeof(int), elementNodePtr.size());
            allocateAndTransfer((void **) &elementAdjNodeIdx_dev, (void *) elementAdjNodeIdx.data(),
                                sizeof(int), elementAdjNodeIdx.size());
        }

        // INIT CUDA PATCH TO PATCH ADJACENT LIST
        int *patchPatchPtr_dev, *patchAdjPatchIdx_dev;
        {
            allocateAndTransfer((void **) &patchPatchPtr_dev, (void *) patchPatchPtr.data(),
                                sizeof(int), patchPatchPtr.size());
            allocateAndTransfer((void **) &patchAdjPatchIdx_dev, (void *) patchAdjPatchIdx.data(),
                                sizeof(int), patchAdjPatchIdx.size());
        }

        // INIT CUDA MPRIMEMATRIX PER ELEMENT LIST
        MprimeMatrix<3> *MprimeList_dev;
        {
            allocateAndTransfer((void **) &MprimeList_dev, (void *) MprimeList.data(),
                                sizeof(MprimeMatrix<3>), MprimeList.size());
        }

        // INIT CUDA TIME REDUCTION LIST AND POINTER LIST
        int *time_reduction_ptr_dev;
        double *time_reduction_list_dev;
        {
            allocateAndTransfer((void **) &time_reduction_ptr_dev, (void *) time_reduction_ptr.data(),
                                sizeof(int), time_reduction_ptr.size());

            // init all time reduction list at MAXF
            int list_size = reduction_span * n_nodes;
            hipMalloc((void **) &time_reduction_list_dev, sizeof(double) * list_size);
            int n_blocks = (list_size + block_size) / block_size;
            initTimeReductionList<<<n_blocks, block_size>>>(time_reduction_list_dev, list_size);
        }

        // INIT CUDA SUPPORT LISTS FOR CONVERGED NODES AND PATCHES
        int *converged_node_list_dev, *converged_patch_list_dev;
        {
            hipMalloc((void **) &converged_node_list_dev, sizeof(int) * n_nodes);
            hipMalloc((void **) &converged_patch_list_dev, sizeof(int) * n_patches);
        }

        // INIT CUDA TIME
        double *U_dev;
        {
            allocateAndTransfer((void **) &U_dev, (void *) U.data(),
                                sizeof(double), U.size());
        }

        // INIT CUDA ACTIVE LIST AND SIZE
        int activeListSize = XPatches.size();
        int *activePatchList_dev, *activePatchListSize_dev;
        {
            allocateAndTransfer((void **) &activePatchList_dev, (void *) XPatches.data(),
                                sizeof(int), XPatches.size());
            allocateAndTransfer((void **) &activePatchListSize_dev, (void *) &activeListSize,
                                sizeof(int), 1);
        }

        while (activeListSize > 0) {

            updatePatchValues3<<<activeListSize, block_size>>>(U_dev, MprimeList_dev, 7,
                                                               activePatchList_dev, activePatchListSize_dev,
                                                               patchElementPtr_dev, patchAdjElementIdx_dev,
                                                               patchNodePtr_dev, patchAdjNodeIdx_dev,
                                                               elementNodePtr_dev, elementAdjNodeIdx_dev,
                                                               time_reduction_list_dev, time_reduction_ptr_dev,
                                                               reduction_span,
                                                               converged_node_list_dev, converged_patch_list_dev);

            activateNeighbours<<<activeListSize, block_size>>>(patchPatchPtr_dev, patchAdjPatchIdx_dev, converged_patch_list_dev,
                                         n_patches);

            scanAndPack<<<1, block_size>>>(converged_patch_list_dev, activePatchList_dev, activePatchListSize_dev, n_patches);
            hipMemcpy(&activeListSize, activePatchListSize_dev, sizeof(int), hipMemcpyDeviceToHost);

            std::cout << "actives" << activeListSize << std::endl;

            updatePatchValues3<<<activeListSize, block_size>>>(U_dev, MprimeList_dev, 1,
                                            activePatchList_dev, activePatchListSize_dev,
                                            patchElementPtr_dev, patchAdjElementIdx_dev,
                                            patchNodePtr_dev, patchAdjElementIdx_dev,
                                            elementNodePtr_dev, elementAdjNodeIdx_dev,
                                            time_reduction_list_dev, time_reduction_ptr_dev, reduction_span,
                                            converged_node_list_dev, converged_patch_list_dev);

            scanAndPack<<<1 , block_size>>>(converged_patch_list_dev, activePatchList_dev, activePatchListSize_dev, n_patches);
            hipMemcpy(&activeListSize, activePatchListSize_dev, sizeof(int), hipMemcpyDeviceToHost);
        }

        hipMemcpy(U.data(), U_dev, sizeof(double) * U.size(), hipMemcpyDeviceToHost);

        // FREE ALL CUDA MEMORY
        {
            hipFree(patchElementPtr_dev);
            hipFree(patchAdjElementIdx_dev);
            hipFree(patchNodePtr_dev);
            hipFree(patchAdjNodeIdx_dev);
            hipFree(elementNodePtr_dev);
            hipFree(elementAdjNodeIdx_dev);
            hipFree(patchPatchPtr_dev);
            hipFree(patchAdjPatchIdx_dev);
            hipFree(activePatchList_dev);
            hipFree(U_dev);
            hipFree(MprimeList_dev);
            hipFree(time_reduction_ptr_dev);
            hipFree(time_reduction_list_dev);
            hipFree(converged_node_list_dev);
            hipFree(converged_patch_list_dev);
            hipFree(activePatchListSize_dev);
        }

        (*result) = true;
    }

    template<>
    void globalSolve<4>(const std::vector<int> &XPatches,
                        const std::vector<int> &patchElementPtr, const std::vector<int> &patchAdjElementIdx,
                        const std::vector<int> &patchNodePtr, const std::vector<int> &patchAdjNodeIdx,
                        const std::vector<int> &elementNodePtr, const std::vector<int> &elementAdjNodeIdx,
                        const std::vector<int> &patchPatchPtr, const std::vector<int> &patchAdjPatchIdx,
                        std::vector<double> &U, const std::vector<MprimeMatrix<4>> &MprimeList,
                        const std::vector<int> &time_reduction_ptr, const int reduction_span,
                        bool *result) {
        return;
    }
}