#include "hip/hip_runtime.h"

#include "LocalSolverKernels.hpp"

namespace Eikonal{

    // EIKONAL MATH KERNELS --------------------------------------------------------------------------------
    __device__ inline bool sameSign(double a, double b){
        return a*b >= 0;
    }

    __device__ inline double
    constraintBisection(double ma, double mb, double mc, double dt, const int max_iters, const double tol) {
        auto f = [ma, mb, mc, dt](const double x) {
            return (x * ma + mb) / std::sqrt(2 * x * mb + x * x * ma + mc) - dt;
        };

        double a = 0;
        double b = 1;
        double fa = f(a);
        double fb = f(b);
        if (sameSign(fa, fb)) {
            if (std::abs(fa) < std::abs(fb))
                return 0;
            else
                return 1;
        }
        double k;
        int iters = max_iters;
        double res;
        do {
            k = (a + b) / 2;
            res = f(k);
            if (sameSign(fa, res)) {
                a = k;
                fa = res;
            } else {
                b = k;
            }
            iters--;
        } while (std::abs(res) > tol && iters != 0);
        return k;
    }
    // -------------------------------------------------------------------------------------------------------



    // GLOBALLY USED LOCAL SOLVER FUNCTIONS ----------------------------------------------------------
    __device__ int esgn(const int &kcode, const int &lcode, const int &scode){
        return (2 * (scode < kcode) - 1) * (2 * (scode < lcode) - 1);
    }

    __device__ int signParity(const int &graycode){
        int s = s - ((s >> 1) & 033333333333) - ((s >> 2) & 011111111111);
        return 1 - 2 * ((((s + (s >> 3)) & 030707070707) % 63) % 2);
    }
    // -----------------------------------------------------------------------------------------------



    // TRIANGULAR LOCAL SOLVER KERNELS --------------------------------------------------------------
    __device__ inline void getSigns3(int *signs, const int &shift){
        signs[0] = signParity((24 >> shift) & 7);
        signs[1] = signParity((16 >> shift) & 7);
    }


    __device__ void getMprimeMatrix3(const int &ptidx, const MprimeMatrix<3> &MT, const double *valin,
                                        MprimeMatrix<3> &M, double *valout){
        constexpr int RED_SIZE = 3 - 1;
        int gcodes[RED_SIZE];
        int signs[RED_SIZE];

        // get gcodes (rotated of -1)
        int refId = (3 + ptidx - 1) % 3;
        int refcode = (1 << refId);

        for (int i = 0; i < RED_SIZE; ++i) {
            int k = (ptidx + i) % 3;
            gcodes[(RED_SIZE + i - 1)%RED_SIZE] = refcode + (1 << k);
        }

        // get rotated vals (rotated of shift)
        int shift = RED_SIZE - ptidx;
        for (int i = 0; i < RED_SIZE; ++i) {
            valout[i] = valin[(3 + i - shift) % 3];
        }

        getSigns3(signs, shift);

        int e0 = gcodes[0] / 2 - 1;
        M(0) = MT(e0);

        int e1 = gcodes[1] / 2 - 1;
        M(1) = MT(e1);

        int g2 = gcodes[0] xor gcodes[1];
        int s01 = esgn(gcodes[0], gcodes[1], g2);
        int e2 = g2 / 2 - 1;
        M(2) = s01 * signs[0] * signs[1] * (MT(e0) + MT(e1) - MT(e2)) / 2;
    }

    __device__ inline double distance3(const double l1, const MprimeMatrix<3> &M) {
        return std::sqrt(l1 * l1 * M(0) + 2 * l1 * M(2) + M(1));
    }


    template<>
    __device__ double solveLocal <3>(const int &pointref, const MprimeMatrix<3> &MT, double *valin){
        TTraits<3>::MprimeMatrix M;
        double valout[3];

        getMprimeMatrix3(pointref, MT, valin, M, valout);

        double t12 = valout[1] - valout[0];
        double l1 = constraintBisection(M(0), M(2), M(1), t12, 5000, 10e-6);
        return -t12 * l1 + valout[1] + distance3(l1, M);
    }

    // ------------------------------------------------------------------------------------------------


    // TETRAHEDRAL LOCAL SOLVER KERNELS ---------------------------------------------------------------
    template<>
    __device__ double solveLocal <4>(const int &pointref, const MprimeMatrix<4> &MT, double *valin){
//        TTraits<3>::MprimeMatrix M;
//        double valout[3];
//
//        getMprimeMatrix3(pointref, MT, valin, M, valout);
//
//        double t12 = valout[1] - valout[0];
//        double l1 = constraintBisection(M(0), M(2), M(1), t12, 5000, 10e-6);
//        return -t12 * l1 + valout[1] + distance3(l1, M);
        return 0;
    }



    // ------------------------------------------------------------------------------------------------
}